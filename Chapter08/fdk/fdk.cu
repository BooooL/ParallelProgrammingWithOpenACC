/*  
******************************************************
This program is to reconstruct for 3-D cone beam projection, apply on 3-D shep-Logan head phaton
There are three steps to the weighted filtered backprojection algorithm: 
1) convert projection to projection_prime (weighted)
2) filtering part
3) backprojection part

reference book: "Principles of Computerized Tomographic Imaging"
                 Avinash C. Kak  & Malcolm Slaney    Page 100-107
implement in Frequency Domain

Permission to use, copy, distribute and modify this software for any 
purpose with or without fee is hereby granted. This software is        
provided "as is" without express or implied warranty. 

Send comments or suggestions for this OpenACC version to
            rxu6@uh.edu, schandra@udel.edu

Authors: Rengan Xu, Sunita Chandrasekaran

May 26th, 2016
******************************************************
*/

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>
#include<string.h>
#include<hip/hip_runtime.h>

#define PI 3.141592653589793
#define SO 62.0  // distance from source to rotation center
#define SD 83.0 // distance from detector to source center. originally 83
#define OD (SO/SD)  // for translate real detector to the dector at origin
#define PROJECTION_Y  200    // number of projections in y axis of detector
#define PROJECTION_Z  200 // number of projections in z axis of detector
#define frame_size (PROJECTION_Z*PROJECTION_Y)
#define Z_CENTER 0  // 115.5
#define Y_CENTER 100 // 515.5
#define sample_interval_y (0.0194*OD)   // sample interval in image detector at origin (not true interval at real detector)
#define sample_interval_z (0.0194*OD)
#define ZP 256  // zero padding
#define RECONSIZE 200
#define RECONSIZE_Z 200
#define REC_XY_CENTER ((RECONSIZE-1.0)/2.0)
#define CONVOLVESIZE ZP/2   // CONVOLVESIZE = ZP/2
#define num_belta 300
#define belta_step (360.0/300.0) //1.0
#define recon_step (sample_interval_y) // voxel size
#define recon_step_z   recon_step
#define water 0.2006
#define ignore1 0
#define open 9013
#define zstart 200  // go up
#define THREADS 128

void four1(float data[], unsigned long nn, int isign);
void realft(float data[], unsigned long n, int isign);
void cosft1(float y[], int n);
//version 2.0
/************************************ Kernel for the 3rd step back projeciton *******************************/
__global__ void back_projection(float *fp_d, short int* CT_numbers_d, int view_start,
					 int view_end, int X_SIZE, int Y_SIZE, int Z_SIZE)
{
	float x,y,z,t,s,p_prime, ksi_prime,SO_s,factor,belta_rad,cos_belta,sin_belta;
	float m_f,m_z,n_f,n_y,temp;
	int m_less,n_less, i, j ,k, l, idx;
	short int CT_number;
	float rec;
    int size2;
    
    idx = threadIdx.x + blockIdx.x*blockDim.x;
    size2 = RECONSIZE*RECONSIZE;
//    size1 = RECONSIZE_Z*size2;

//for(idx=idx; idx<size1; idx+=blockDim.x*gridDim.x)
{
    i = (idx/size2)%RECONSIZE_Z;
    j = (idx/RECONSIZE)%RECONSIZE;
    k = idx%RECONSIZE;
	
    z=(Z_CENTER-zstart+i)*recon_step_z;
	y=(j-REC_XY_CENTER)*recon_step;
	x=(k-REC_XY_CENTER)*recon_step;
	rec = 0;
	for(l=view_start;l<view_end;l++)  
	{     
		belta_rad=(num_belta-l)*belta_step*PI/180;
		cos_belta=cos(belta_rad); sin_belta=sin(belta_rad);
		t=x*cos_belta+y*sin_belta;
		s=y*cos_belta-x*sin_belta;
		SO_s=SO/(SO-s);
		p_prime=SO_s*t;
		ksi_prime=SO_s*z;
		factor=SO_s*SO_s;
		/*    bilinear interpolation  */
		m_f=-ksi_prime/sample_interval_z+Z_CENTER;
		m_less=(int)floor(m_f);
		m_z=(m_f-m_less);
		n_f=p_prime/sample_interval_y+Y_CENTER;
		n_less=(int)floor(n_f);
		n_y=(n_f-n_less);
					
        if (m_less>=199) m_less = PROJECTION_Z-2;
        if (n_less>=255) n_less = ZP-2;
        if (m_less<=0) m_less = 0;
        if (n_less<=0) n_less = 0;

		temp=(1-m_z)*(1-n_y)*fp_d[l*Y_SIZE*Z_SIZE+m_less*Z_SIZE+n_less]+m_z*(1-n_y)*fp_d[l*Y_SIZE*Z_SIZE+(m_less+1)*Z_SIZE+n_less]+
			(1-m_z)*n_y*fp_d[l*Y_SIZE*Z_SIZE+m_less*Z_SIZE+n_less+1]+m_z*n_y*fp_d[l*Y_SIZE*Z_SIZE+(m_less+1)*Z_SIZE+n_less+1];

		rec+=factor*temp;
	}   // end of belta--viewend
	temp=rec*4*PI/num_belta;
	if(temp<0) 
		temp=0.0;
	CT_number=(short int)((temp-water)/water*1000);
	CT_numbers_d[i*RECONSIZE*RECONSIZE + j*RECONSIZE + k] = CT_number;
  }
}

/*************************************               MAIN    *************************************************/

int main(int argc, char** argv)  
{
    if(argc < 3)
    {
        printf("usage: %s <input> <output>\n", argv[0]);
        exit(0);
    }
	
    FILE *ptr_proj,*ptr_ct;

	int i,j,k,l,n,skip;
	static unsigned short proj;
	static float projection[ZP];
	static float weight[PROJECTION_Z][PROJECTION_Y];

    // y_prime & z_prime are coordinate in detector; y[] & z[] are coordinate in detector of rotation center
	float y_prime,z_prime,filter[CONVOLVESIZE+1];  

	int num_view,view_start,view_end;

	float *fp_h;
	float *fp_d;
	short int *CT_numbers_h;
	short int *CT_numbers_d;
	float ***filteredprojection;
    
    struct timeval tim;
    double begin, end;
    

	int X_SIZE = 300;
	int Y_SIZE = PROJECTION_Z;
	int Z_SIZE = ZP;
	filteredprojection = (float ***)malloc(sizeof(float **) * X_SIZE);
	 
	for (i = 0 ;  i < X_SIZE; i++) {
	   filteredprojection[i] = (float **)malloc(sizeof(float *) * Y_SIZE);
	 
	   for (j = 0; j < Y_SIZE; j++)
		  filteredprojection[i][j] = (float *)malloc(sizeof(float) * Z_SIZE);
	}

	// ramp filter design
	for(n=0;n<CONVOLVESIZE+1;n++)  
	{
		if(n==0) 
			filter[n]=1/(8*(sample_interval_y)*(sample_interval_y));
		else  
			if((n%2)==0) 
				filter[n]=0;
			else 
				filter[n]=-1/(2*n*n*PI*PI*(sample_interval_y)*(sample_interval_y));
	}
	cosft1(filter-1,CONVOLVESIZE);  // FFT

	for(i=0;i<PROJECTION_Z;i++)  
	{   // weitht factor is independent of belta rotation angle
		z_prime=-(i-Z_CENTER)*sample_interval_z;  // z center 414
		for(j=0;j<PROJECTION_Y;j++)  
		{
			y_prime=(j-Y_CENTER)*sample_interval_y;   // y center 504
			weight[i][j]=SO/sqrt(SO*SO+ y_prime *y_prime+z_prime*z_prime);
		}
	}
	
	num_view=num_belta;//process_size;
	view_start=0; //my_rank * num_view;
	view_end=view_start+num_view;

	if((ptr_proj=fopen(argv[1],"rb"))==NULL)  
	{ // If file open is not succesful, print could not open the file and quit
		fprintf(stderr,"Sorry could not open the file %s.\n", argv[1]);
		exit(1);
	}

	if((ptr_ct=fopen(argv[2],"wb"))==NULL ) 
	{ // If file open is not succesful, print could not open the file and quit
		fprintf(stderr,"Sorry could not open the file %s.\n", argv[2]);
		exit(1);
	}
    
    gettimeofday(&tim, NULL);
    begin = tim.tv_sec + (tim.tv_usec/1000000.0);

	// filtering projection data of each theta angle  in frequency domain
	for(l=view_start;l<view_end;l++)  
	{  // start of l ----------------
		skip=num_belta-l+ignore1;
		fseek(ptr_proj,(skip*frame_size)*sizeof(unsigned short),0);  // skip seq header , start from current frame l

		//step 1: convert projection to projection_prime  page 106 function (175)
		for(i=0;i<PROJECTION_Z;i++)  
		{
			for(j=0;j<ZP;j++)  
			{
				if( j<PROJECTION_Y)  
				{
					fread(&proj,sizeof(unsigned short),1,ptr_proj);
					//  if(j<50 || j>1900) proj=open; //correction for the collimators
					if( proj==0 )  
						proj=open;
					if(proj>=open) 
						projection[j]=0;
					else 
						projection[j]=-(log(proj*1.0/open))*weight[i][j];
				}
				else 
					projection[j]=0;    // zero padding
			}  //end of j-PROJECTION_Y
			realft(projection-1,ZP,1);       //  FFT

		   //step 2: filter projection
			for(j=0;j<ZP;j++)  
			{  // filter process in freq domain == filter * projection
				if((j%2)==0)  
					filteredprojection[l][i][j]=projection[j]*filter[j/2]*(sample_interval_y)*2/ZP;
				else  
					filteredprojection[l][i][j]=projection[j]*filter[(j-1)/2]*(sample_interval_y)*2/ZP;
			}
			realft(filteredprojection[l][i]-1,ZP,-1);    // IFFT
		}  // end of i-PROJECTION_Z
		
	 } // end of belta
    

	//step 3: back projection

	fp_h = (float*)malloc(X_SIZE * Y_SIZE * Z_SIZE * sizeof (float));
	for( i=0; i<X_SIZE; i++ )
		for( j=0; j<Y_SIZE; j++)
			for( k=0; k<Z_SIZE; k++)
			{
				fp_h[i*Y_SIZE*Z_SIZE+j*Z_SIZE+k] = filteredprojection[i][j][k];
			}
	 
	hipMalloc((void**)&fp_d, X_SIZE * Y_SIZE * Z_SIZE * sizeof (float));
	hipMemcpy(fp_d, fp_h, X_SIZE * Y_SIZE * Z_SIZE * sizeof (float), hipMemcpyHostToDevice);
	
	hipMalloc((void**)&CT_numbers_d, RECONSIZE_Z * RECONSIZE * RECONSIZE * sizeof (short int));
	hipMemset((void*)CT_numbers_d, 0, RECONSIZE_Z * RECONSIZE * RECONSIZE * sizeof (short int));
	CT_numbers_h = (short int*)malloc(RECONSIZE_Z * RECONSIZE * RECONSIZE * sizeof (short int));
	
	dim3 dimBlock(THREADS, 1, 1);
	dim3 dimGrid((RECONSIZE_Z*RECONSIZE*RECONSIZE+THREADS-1)/THREADS, 1, 1);
    
	back_projection<<<dimGrid, dimBlock>>>(fp_d, CT_numbers_d, view_start, view_end, X_SIZE, Y_SIZE, Z_SIZE);

	hipMemcpy(CT_numbers_h, CT_numbers_d, RECONSIZE_Z * RECONSIZE * RECONSIZE * sizeof (short int), hipMemcpyDeviceToHost);
    
    gettimeofday(&tim, NULL);
    end = tim.tv_sec + (tim.tv_usec/1000000.0);
    
	fwrite(CT_numbers_h,sizeof(short int),RECONSIZE_Z * RECONSIZE * RECONSIZE,ptr_ct);
	
	fclose(ptr_proj);
    fclose(ptr_ct);
	free(fp_h);
	free(CT_numbers_h);
    

	printf("Execution time of FDK: %.2f seconds\n",end-begin);
} // end of main

/*********************************   FFT   ******************************************/
#define SWAP(a,b) tempr=(a);(a)=(b);(b)=tempr

void four1(float data[], unsigned long nn, int isign)
{
	unsigned long n,mmax,m,j,istep,i;
	double wtemp,wr,wpr,wpi,wi,theta;
	float tempr,tempi;

	n=nn << 1;
	j=1;
	for (i=1;i<n;i+=2) 
	{
		if (j > i) 
		{
			SWAP(data[j],data[i]);
			SWAP(data[j+1],data[i+1]);
		}
		m=n >> 1;
		while (m >= 2 && j > m) 
		{
			j -= m;
			m >>= 1;
		}
		j += m;
	}
	mmax=2;
	while (n > mmax) 
	{
		istep=mmax << 1;
		theta=isign*(6.28318530717959/mmax);
		wtemp=sin(0.5*theta);
		wpr = -2.0*wtemp*wtemp;
		wpi=sin(theta);
		wr=1.0;
		wi=0.0;
		for (m=1;m<mmax;m+=2) 
		{
			for (i=m;i<=n;i+=istep) 
			{
				j=i+mmax;
				tempr=wr*data[j]-wi*data[j+1];
				tempi=wr*data[j+1]+wi*data[j];
				data[j]=data[i]-tempr;
				data[j+1]=data[i+1]-tempi;
				data[i] += tempr;
				data[i+1] += tempi;
			}
			wr=(wtemp=wr)*wpr-wi*wpi+wr;
			wi=wi*wpr+wtemp*wpi+wi;
		}
		mmax=istep;
	}
}
#undef SWAP

void realft(float data[], unsigned long n, int isign)
{
	void four1(float data[], unsigned long nn, int isign);
	unsigned long i,i1,i2,i3,i4,np3;
	float c1=0.5,c2,h1r,h1i,h2r,h2i;
	double wr,wi,wpr,wpi,wtemp,theta;

	theta=3.141592653589793/(double) (n>>1);
	if (isign == 1) 
	{
		c2 = -0.5;
		four1(data,n>>1,1);
	} 
	else 
	{
		c2=0.5;
		theta = -theta;
	}
	wtemp=sin(0.5*theta);
	wpr = -2.0*wtemp*wtemp;
	wpi=sin(theta);
	wr=1.0+wpr;
	wi=wpi;
	np3=n+3;
	for (i=2;i<=(n>>2);i++) 
	{
		i4=1+(i3=np3-(i2=1+(i1=i+i-1)));
		h1r=c1*(data[i1]+data[i3]);
		h1i=c1*(data[i2]-data[i4]);
		h2r = -c2*(data[i2]+data[i4]);
		h2i=c2*(data[i1]-data[i3]);
		data[i1]=h1r+wr*h2r-wi*h2i;
		data[i2]=h1i+wr*h2i+wi*h2r;
		data[i3]=h1r-wr*h2r+wi*h2i;
		data[i4] = -h1i+wr*h2i+wi*h2r;
		wr=(wtemp=wr)*wpr-wi*wpi+wr;
		wi=wi*wpr+wtemp*wpi+wi;
	}
	if (isign == 1) 
	{
		data[1] = (h1r=data[1])+data[2];
		data[2] = h1r-data[2];
	} 
	else 
	{
		data[1]=c1*((h1r=data[1])+data[2]);
		data[2]=c1*(h1r-data[2]);
		four1(data,n>>1,-1);
	}
}

void cosft1(float y[], int n)
{
	void realft(float data[], unsigned long n, int isign);
	int j,n2;
	float sum,y1,y2;
	double theta,wi=0.0,wpi,wpr,wr=1.0,wtemp;

	theta=PI/n;
	wtemp=sin(0.5*theta);
	wpr = -2.0*wtemp*wtemp;
	wpi=sin(theta);
	sum=0.5*(y[1]-y[n+1]);
	y[1]=0.5*(y[1]+y[n+1]);
	n2=n+2;
	for (j=2;j<=(n>>1);j++) 
	{
		wr=(wtemp=wr)*wpr-wi*wpi+wr;
		wi=wi*wpr+wtemp*wpi+wi;
		y1=0.5*(y[j]+y[n2-j]);
		y2=(y[j]-y[n2-j]);
		y[j]=y1-wi*y2;
		y[n2-j]=y1+wi*y2;
		sum += wr*y2;
	}
	realft(y,n,1);
	y[n+1]=y[2];
	y[2]=sum;
	for (j=4;j<=n;j+=2) 
	{
		sum += y[j];
		y[j]=sum;
	}
}

#undef PI
