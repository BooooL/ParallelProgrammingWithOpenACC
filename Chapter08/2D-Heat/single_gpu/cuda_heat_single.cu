/*
******************************************************
This file is the single GPU version of 2D Heat Equation 
using CUDA programming model. This implementation is based
on the CPU version from
http://www.many-core.group.cam.ac.uk/archive/CUDAcourse09/

Permission to use, copy, distribute and modify this software for any 
purpose with or without fee is hereby granted. This software is        
provided "as is" without express or implied warranty. 

Send comments or suggestions for this OpenACC version to
            rxu6@uh.edu, schandra@udel.edu

Authors: Rengan Xu, Sunita Chandrasekaran

May 26th, 2016
******************************************************
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <sys/time.h>
#include <fcntl.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

// I2D to index into a linear memory space from a 2D array index pair
#define I2D(ni, i, j) ((i) + (ni)*(j))
#define THREADS 128

// kernel to update temperatures - CPU version
__global__ void step_kernel_cpu(int ni, 
                     int nj,
                     double tfac, 
                     double *temp_in,
                     double *temp_out) {
    int i, j, i00, im10, ip10, i0m1, i0p1;
    double d2tdx2, d2tdy2;

    j = blockIdx.y + 1;
    i = threadIdx.x + blockIdx.x*blockDim.x + 1;

    // loop over all points in domain (not boundary points)
    for (j=j; j < nj-1; j+=gridDim.y) {
        for (i=i; i < ni-1; i+=blockDim.x*gridDim.x) {
	    // find indices into linear memory for central point and neighbours
            i00 = I2D(ni, i, j);
            im10 = I2D(ni, i-1, j);
            ip10 = I2D(ni, i+1, j);
            i0m1 = I2D(ni, i, j-1);
            i0p1 = I2D(ni, i, j+1);

	    // evaluate derivatives
            d2tdx2 = temp_in[im10] - 2*temp_in[i00] + temp_in[ip10];
            d2tdy2 = temp_in[i0m1] - 2*temp_in[i00] + temp_in[i0p1];

	    // update temperatures
            temp_out[i00] = temp_in[i00] + tfac*(d2tdx2 + d2tdy2);
        }
    }

}


int main(int argc, char *argv[]) 
{
    
    if(argc < 5)
    {
        printf("Usage: %s <ni> <ni> <nstep> <output file>\n", argv[0]);
        exit(0);
    }

    int ni, nj, nstep;
    double tfac, *temp1_h, *temp2_h, *temp_tmp, *temp1_d, *temp2_d;
    int i, j, i2d, istep;
    double temp_bl, temp_br, temp_tl, temp_tr;
	struct timeval tim;
	double start, end;
	int fd;
   
    // domain size and number of timesteps (iterations)
    ni = atoi(argv[1]);
    nj = atoi(argv[2]);
    nstep = atoi(argv[3]);
    
    // allocate temperature array on host
    temp1_h = (double *)malloc(sizeof(double)*(ni+2)*(nj+2));
    temp2_h = (double *)malloc(sizeof(double)*(ni+2)*(nj+2));

    // initial temperature in interior
    for (j=1; j < nj+1; j++) {
        for (i=1; i < ni+1; i++) {
            i2d = i + (ni+2)*j;
            temp1_h[i2d] = 0.0;
        }
    }
    
    // initial temperature on boundaries - set corners
    temp_bl = 200.0f;
    temp_br = 300.0f;
    temp_tl = 200.0f;
    temp_tr = 300.0f;

    // set edges by linear interpolation from corners
    for (i=0; i < ni+2; i++) {
        // bottom
        j = 0;
        i2d = i + (ni+2)*j;
        temp1_h[i2d] = temp_bl + (temp_br-temp_bl)*(double)i/(double)(ni+1);

        // top
        j = nj+1;
        i2d = i + (ni+2)*j;
        temp1_h[i2d] = temp_tl + (temp_tr-temp_tl)*(double)i/(double)(ni+1);
    }

    for (j=0; j < nj+2; j++) {
        // left
        i = 0;
        i2d = i + (ni+2)*j;
        temp1_h[i2d] = temp_bl + (temp_tl-temp_bl)*(double)j/(double)(nj+1);

        // right
        i = ni+1;
        i2d = i + (ni+2)*j;
        temp1_h[i2d] = temp_br + (temp_tr-temp_br)*(double)j/(double)(nj+1);
    }

    // duplicate temeperature array on host
    memcpy(temp2_h, temp1_h, sizeof(double)*(ni+2)*(nj+2));
    
	gettimeofday(&tim, NULL);
	start = tim.tv_sec + (tim.tv_usec/1000000.0);
	
    // allocate temperature arrays on device
    hipMalloc((void **)&temp1_d, sizeof(double)*(ni+2)*(nj+2));
    hipMalloc((void **)&temp2_d, sizeof(double)*(ni+2)*(nj+2));
    
    // transfer temperature array from host to device
    hipMemcpy((void *)temp1_d, (void *)temp1_h, sizeof(double)*(ni+2)*(nj+2),
               hipMemcpyHostToDevice);
    hipMemcpy((void *)temp2_d, (void *)temp1_h, sizeof(double)*(ni+2)*(nj+2),
               hipMemcpyHostToDevice);

    tfac = 0.2;
    
    // main iteration loop  

    for (istep=0; istep < nstep; istep++) {
            dim3 grid_dim((ni+THREADS-1)/THREADS, nj, 1); 
            dim3 block_dim(THREADS, 1, 1); 
            step_kernel_cpu<<<grid_dim, block_dim>>>(ni+2, nj+2, tfac, temp1_d, temp2_d);

	    // swap the temp pointers
            temp_tmp = temp1_d;
            temp1_d = temp2_d;
            temp2_d = temp_tmp;
            
    } 
    
    hipMemcpy((void *)temp1_h, (void *)temp1_d, sizeof(double)*(ni+2)*(nj+2),
                   hipMemcpyDeviceToHost);
	
    gettimeofday(&tim, NULL);
    end = tim.tv_sec + (tim.tv_usec/1000000.0);
    printf("Time for computing: %.2f s\n",end-start);


    // output temp1 to a file
    
	fd = creat(argv[4], 00666);
	fd = open(argv[4], O_WRONLY);
	write(fd, temp1_h, (size_t)(ni+2)*(nj+2)*sizeof(double));
	close(fd);
    
/*
    FILE *fp;
    fp = fopen(filename, "w");
    fprintf(fp, "%d %d\n", ni, nj);
    for (j=0; j < nj; j++) {
        for (i=0; i < ni; i++) {
            fprintf(fp, "%.4f\n", j, i, temp1_h[i + ni*j]);
        }
    }
    fclose(fp);
*/
}


