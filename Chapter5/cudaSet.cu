/****************************************
ADD Description
TODO
*****************************************/


#include <hip/hip_runtime.h>

__global__ void setVal(double * B, size_t size, double val)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(; tid < size; tid += stride)
        B[tid] = val;
}

extern "C" 
{
void cudaSet(double * B, size_t size, double val) {
    setVal<<<1,128>>>(B,size,val);
}
}
